#include <algorithm>
#include <hip/hip_runtime.h>
#include <iostream>
#include <random>
#include <vector>

__device__ __forceinline__ int binary_search(const int *superList,
                                             const int superList_size,
                                             const int target) {
  int left = 0;
  int right = superList_size - 1;

  while (left <= right) {
    int mid = left + (right - left) / 2;

    if (superList[mid] == target) {
      return mid;
    }
    if (superList[mid] < target) {
      left = mid + 1;
    } else {
      right = mid - 1;
    }
  }
  return -1;
}

__global__ void find_indices(const int *__restrict__ superList,
                             const int superList_size,
                             const int *__restrict__ subList,
                             const int subList_size, int *__restrict__ output,
                             const int *__restrict__ original_indices) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx < subList_size) {
    output[original_indices[idx]] =
        binary_search(superList, superList_size, subList[idx]);
  }
}

int main() {
  int superList_size = 500000000;
  int subList_size = 100000;

  std::vector<int> superList(superList_size);
  std::iota(superList.begin(), superList.end(), 1);

  std::vector<int> subList(subList_size);
  for (int i = 0; i < subList_size; ++i) {
    subList[i] = rand() % superList_size + 1;
  }
  std::cout << "Sublist size: " << subList.size() << std::endl;

  std::vector<int> indices(subList_size);
  std::iota(indices.begin(), indices.end(), 0);
  std::sort(indices.begin(), indices.end(),
            [&subList](int a, int b) { return subList[a] < subList[b]; });

  std::vector<int> sorted_subList(subList_size);
  for (int i = 0; i < subList_size; ++i) {
    sorted_subList[i] = subList[indices[i]];
  }

  int *d_superList, *d_subList, *d_output, *d_indices;
  int *h_output;
  hipHostMalloc(&h_output, subList_size * sizeof(int), hipHostMallocDefault);

  hipMalloc(&d_superList, superList_size * sizeof(int));
  hipMalloc(&d_subList, subList_size * sizeof(int));
  hipMalloc(&d_output, subList_size * sizeof(int));
  hipMalloc(&d_indices, subList_size * sizeof(int));

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipMemcpy(d_superList, superList.data(), superList_size * sizeof(int),
             hipMemcpyHostToDevice);
  hipMemcpy(d_subList, sorted_subList.data(), subList_size * sizeof(int),
             hipMemcpyHostToDevice);
  hipMemcpy(d_indices, indices.data(), subList_size * sizeof(int),
             hipMemcpyHostToDevice);

  int blockSize = 512;
  int numBlocks = (subList_size + blockSize - 1) / blockSize;

  hipEventRecord(start);

  find_indices<<<numBlocks, blockSize>>>(d_superList, superList_size, d_subList,
                                         subList_size, d_output, d_indices);

  hipEventRecord(stop);
  hipEventSynchronize(stop);

  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);

  hipMemcpy(h_output, d_output, subList_size * sizeof(int),
             hipMemcpyDeviceToHost);

  std::cout << "Kernel execution time: " << milliseconds << " ms" << std::endl;

  for (int i = 0; i < subList_size; ++i) {
    if (superList[h_output[i]] != subList[i]) {
      std::cout << "Mismatch at index " << h_output[i] << std::endl;
      std::cout << "Expected: " << superList[h_output[i]]
                << ", got: " << h_output[i] << std::endl;
      break;
    }
  }

  hipFree(d_superList);
  hipFree(d_subList);
  hipFree(d_output);
  hipFree(d_indices);
  hipHostFree(h_output);
  hipEventDestroy(start);
  hipEventDestroy(stop);

  return 0;
}