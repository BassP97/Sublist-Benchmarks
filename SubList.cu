#include <algorithm>
#include <hip/hip_runtime.h>
#include <iostream>
#include <random>
#include <vector>

__device__ int binary_search(const int *superList, int superList_size,
                             int target) {
  int left = 0;
  int right = superList_size - 1;
  while (left <= right) {
    int mid = left + (right - left) / 2;
    if (superList[mid] == target) {
      return mid;
    } else if (superList[mid] < target) {
      left = mid + 1;
    } else {
      right = mid - 1;
    }
  }
  return -1;
}

__global__ void find_indices(const int *superList, int superList_size,
                             const int *subList, int subList_size,
                             int *output) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < subList_size) {
    output[idx] = binary_search(superList, superList_size, subList[idx]);
  }
}

int main() {
  std::vector<int> superList(500000000);
  std::iota(superList.begin(), superList.end(), 1);

  std::vector<int> subList(10000);
  for (int i = 0; i < 10000; ++i) {
    subList[i] = rand() % 500000000 + 1;
  }

  int superList_size = superList.size();
  int subList_size = subList.size();

  int *d_superList;
  int *d_subList;
  int *d_output;
  int *h_output = new int[subList_size];

  hipMalloc(&d_superList, superList_size * sizeof(int));
  hipMalloc(&d_subList, subList_size * sizeof(int));
  hipMalloc(&d_output, subList_size * sizeof(int));

  hipMemcpy(d_superList, superList.data(), superList_size * sizeof(int),
             hipMemcpyHostToDevice);
  hipMemcpy(d_subList, subList.data(), subList_size * sizeof(int),
             hipMemcpyHostToDevice);

  int blockSize = 256;
  int numBlocks = (subList_size + blockSize - 1) / blockSize;

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);

  find_indices<<<numBlocks, blockSize>>>(d_superList, superList_size, d_subList,
                                         subList_size, d_output);

  hipEventRecord(stop);

  hipEventSynchronize(stop);

  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);

  hipMemcpy(h_output, d_output, subList_size * sizeof(int),
             hipMemcpyDeviceToHost);

  std::cout << "Kernel execution time: " << milliseconds << " ms" << std::endl;

  hipFree(d_superList);
  hipFree(d_subList);
  hipFree(d_output);
  delete[] h_output;

  hipEventDestroy(start);
  hipEventDestroy(stop);

  return 0;
}